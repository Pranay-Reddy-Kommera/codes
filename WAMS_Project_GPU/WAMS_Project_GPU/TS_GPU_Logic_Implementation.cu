#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "TS_Data_GPU.h"


bool HandleCUDAErrors(hipError_t t)
{
	if (t != hipSuccess)
	{
		puts(hipGetErrorString(hipGetLastError()));
		return false;
	}
	return true;
}

bool GetCUDARunTimeError()
{
	hipError_t t = hipGetLastError();
	if (t != hipSuccess)
	{
		puts(hipGetErrorString(t));
		return false;
	}
	return true;
}

__global__ void ProductPMU(float* angle, float* power, float* prod)
{
	unsigned int x = threadIdx.x + (threadIdx.y*blockDim.x);
	unsigned int y = (blockDim.x*blockDim.y*blockIdx.x) + x;

	prod[y] = angle[y] * power[y];

}

float CenterOfAnglesGPU(Performance& pm, float *angle, float *power, float *CenterOfAngle, float *product_PMU)
{
	
	float *angleGPU, *powerGPU, *productGPU, add=0.0, powerAdd=0.0,elapsedTime=0.0;

	//Allocate memory on GPU
	HandleCUDAErrors(hipMalloc((void**)&angleGPU, VECTOR_SIZE_IN_BYTES_PMU));
	HandleCUDAErrors(hipMalloc((void**)&powerGPU, VECTOR_SIZE_IN_BYTES_PMU));
	HandleCUDAErrors(hipMalloc((void**)&productGPU, VECTOR_SIZE_IN_BYTES_PMU));

	pm.OnSetStartTime();
	//Copy data into device memory
	HandleCUDAErrors(hipMemcpy(angleGPU, angle, VECTOR_SIZE_IN_BYTES_PMU, hipMemcpyHostToDevice));
	HandleCUDAErrors(hipMemcpy(powerGPU, power, VECTOR_SIZE_IN_BYTES_PMU, hipMemcpyHostToDevice));
	

	//Grid and Block Setup
	int blockx = 16;
	int blocky = 16;
	dim3 block(blockx, blocky);
	dim3 grid((TOTAL_PMU+255)/256, 1);

	pm.OnSetStopTime();
	float time_Copy = pm.OnGetElapsedTime();
	//cout << "The copy time is " << time_Copy << endl;

	hipEvent_t kernel_start; //Set the event corresponding to kernel lanuch
	hipEvent_t kernel_stop; //Set the event corresponding to kernel completion

	HandleCUDAErrors(hipEventCreate(&kernel_start));//start event object is created here
	HandleCUDAErrors(hipEventCreate(&kernel_stop));//stop event object is created here

	HandleCUDAErrors(hipEventRecord(kernel_start, 0));//recording the kernel start, 0 is a string//its a non blocking call

	//Kernel Calling
	ProductPMU << <grid, block >> >(angleGPU, powerGPU, productGPU);
	//hipDeviceSynchronize();
	GetCUDARunTimeError();

	HandleCUDAErrors(hipEventRecord(kernel_stop, 0));//record the kernel stop
	HandleCUDAErrors(hipEventSynchronize(kernel_stop));//This is a blocking call same as hipDeviceSynchronize

	//Used to store elapsed time
	HandleCUDAErrors(hipEventElapsedTime(&elapsedTime, kernel_start, kernel_stop));//address is where we store the value of elapsed time

	//cout << "The kernel time is" << elapsedTime << endl;
	pm.OnSetStartTime();
	HandleCUDAErrors(hipMemcpy(product_PMU, productGPU, VECTOR_SIZE_IN_BYTES_PMU, hipMemcpyDeviceToHost));

	for (int i = 0; i < TOTAL_PMU; i++)
	{
		add += product_PMU[i];
		powerAdd += power[i];
	}

	CenterOfAngle[0] = add / powerAdd;
	pm.OnSetStopTime();
	float time_CopyBack = pm.OnGetElapsedTime();
	//cout << "The copy back time is " << time_CopyBack << endl;

	float time_Total = (time_Copy * 1000) + elapsedTime + (time_CopyBack * 1000);
	//cout << "The total time is " << time_Total<< endl;
	//cout << "The center of angle is " << CenterOfAngle[0] << endl;

	HandleCUDAErrors(hipFree(angleGPU));
	HandleCUDAErrors(hipFree(powerGPU));
	HandleCUDAErrors(hipFree(productGPU));

	//Remove the kernel events
	HandleCUDAErrors(hipEventDestroy(kernel_start));
	HandleCUDAErrors(hipEventDestroy(kernel_stop));
	HandleCUDAErrors(hipDeviceReset());

	return time_Total;
}


__global__ void Stability(float* d_Angle, float positiveThreshold, float negativeThreshold)
{
	unsigned int x = threadIdx.x + (threadIdx.y*blockDim.x);
	unsigned int y = (blockDim.x*blockDim.y*blockIdx.x) + x;
	//float neg = negativeThreshold;
	//float pos = positiveThreshold;
	
		int temp = 0;
		float area = 0.0;

		for (int i = 0; i < (SAMPLES - 1); i++)
		{
			float anglePrevious = d_Angle[(i*TOTAL_PMU) + y];
			float angle = d_Angle[((i+1)*TOTAL_PMU)+y];
			float diffAngles = angle - anglePrevious;


			if ((angle <= positiveThreshold) && (angle >= negativeThreshold))
			{
				temp += 0;
			}
			else
			{
				if (angle > positiveThreshold)
				{
					if (anglePrevious <= positiveThreshold)
					{
						anglePrevious = positiveThreshold;
					}

					//area += (fabsf(anglePrevious)*TIMESTEP) + (0.5*TIMESTEP*fabsf(diffAngles));

					if (diffAngles < 0)
					{
					diffAngles = -diffAngles;
					}
					if (anglePrevious < 0)
					{
					anglePrevious = -anglePrevious;
					}
					area += (anglePrevious*TIMESTEP) + (0.5*TIMESTEP*(diffAngles));
					if (area >= 5)
					{
						temp += 1;
					}
					else
					{
						temp += 0;
						area = 0.0;
					}
				}
				else
				{
					if (anglePrevious >= negativeThreshold)
					{
						anglePrevious = negativeThreshold;
					}

					//area += (fabsf(anglePrevious)*TIMESTEP) + (0.5*TIMESTEP*fabsf(diffAngles));

					if (diffAngles < 0)
					{
					diffAngles = -diffAngles;
					}
					if (anglePrevious < 0)
					{
					anglePrevious = -anglePrevious;
					}
					area += (anglePrevious*TIMESTEP) + (0.5*TIMESTEP*(diffAngles));
					if (area >= 5)
					{
						temp += 1;
					}
					else
					{
						temp += 0;
						area = 0.0;
					}
				}
			}
		}
		if ((temp > 0)&&(y<TOTAL_PMU))
		{
			//cout << "The system is approaching unstable consition at generator bus number " << j << endl;
			printf("The system is approaching unstable condition at generator bus number %d\n", y);
		}
	
}

float StabilityGPU(Performance& pm, float *angle, float *COA)
{
	float *d_Angle, elapsedTime=0.0;

	//Allocate device memory
	HandleCUDAErrors(hipMalloc((void**)&d_Angle, VECTOR_SIZE_IN_BYTES));

	//Copy data to GPU
	pm.OnSetStartTime();
	HandleCUDAErrors(hipMemcpy(d_Angle, angle, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice));
	

	//Grid and Block Setup
	int blockx = 16;
	int blocky = 16;
	dim3 block(blockx, blocky);
	dim3 grid((TOTAL_PMU + 255) / 256, 1);

	float positiveThreshold = COA[0] + 60;
	float negativeThreshold = COA[0] - 70;

	pm.OnSetStopTime();
	float time_Copy_Stability = pm.OnGetElapsedTime();
	//cout << "The copy time is " << time_Copy_Stability << endl;

	//cout << "The +ve angle is " << positiveThreshold << endl;
	//cout << "The -ve threshold is " << negativeThreshold << endl;
	
	hipEvent_t kernel_start; //Set the event corresponding to kernel lanuch
	hipEvent_t kernel_stop; //Set the event corresponding to kernel completion

	HandleCUDAErrors(hipEventCreate(&kernel_start));//start event object is created here
	HandleCUDAErrors(hipEventCreate(&kernel_stop));//stop event object is created here

	HandleCUDAErrors(hipEventRecord(kernel_start, 0));//recording the kernel start, 0 is a string//its a non blocking call

	Stability << <grid, block >> >(d_Angle,positiveThreshold, negativeThreshold);
	//hipDeviceSynchronize();
	GetCUDARunTimeError();

	HandleCUDAErrors(hipEventRecord(kernel_stop, 0));//record the kernel stop
	HandleCUDAErrors(hipEventSynchronize(kernel_stop));//This is a blocking call same as hipDeviceSynchronize

	//Used to store elapsed time
	HandleCUDAErrors(hipEventElapsedTime(&elapsedTime, kernel_start, kernel_stop));//address is where we store the value of elapsed time
	//cout << "The elapsed time is " << elapsedTime << endl;
	float total_Time = (time_Copy_Stability * 1000) + elapsedTime;
	HandleCUDAErrors(hipFree(d_Angle));

	//Remove the kernel events
	HandleCUDAErrors(hipEventDestroy(kernel_start));
	HandleCUDAErrors(hipEventDestroy(kernel_stop));
	HandleCUDAErrors(hipDeviceReset());

	return total_Time;
}
