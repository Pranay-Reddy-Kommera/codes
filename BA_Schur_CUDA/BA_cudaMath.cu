#include "hip/hip_runtime.h"
#include "BA_data.h"
#include "BA_includeFiles.h"

__global__ void invVcuda(dtype* V_d, int num_pt)
{
	unsigned int k = threadIdx.x + (blockDim.x * threadIdx.y) + (blockDim.x * blockDim.y * blockIdx.x);
	
	if (k <= num_pt){
		dtype a, b, c, d, e, f, g, h, i, det; int Id = k * 9;
		a = V_d[Id]; b = V_d[Id + 1]; c = V_d[Id + 2];
		d = V_d[Id + 3]; e = V_d[Id + 4]; f = V_d[Id + 5];
		g = V_d[Id + 6]; h = V_d[Id + 7]; i = V_d[Id + 8];
		
		det = (a*((e*i) - (h*f))) - (b*((d*i) - (g*f))) + (c*((d*h) - (e*g)));

		V_d[Id] = ((e*i) - (h*f)) / det; V_d[Id + 1] = ((h*c) - (b*i)) / det; V_d[Id + 2] = ((b*f) - (e*c)) / det;
		V_d[Id + 3] = ((f*g) - (d*i)) / det; V_d[Id + 4] = ((a*i) - (g*c)) / det; V_d[Id + 5] = ((d*c) - (a*f)) / det;
		V_d[Id + 6] = ((d*h) - (e*g)) / det; V_d[Id + 7] = ((b*g) - (a*h)) / det; V_d[Id + 8] = ((a*e) - (b*d)) / det;
	}
}

//Kernel
void inverseVCalculation(dtype* V_d)
{
	//Call GPU kernel to perform V inverse
	int dimx = 32; int dimy = 32;
	cout << "Number of blocks is " << (num_pt / (dimx*dimy)) + 1 << endl;
	dim3 block(dimx, dimy); dim3 grid(((num_pt / (block.x*block.y)) + 1));

	invVcuda <<<grid, block >>>(V_d, num_pt);
	hipDeviceSynchronize();
}

__global__ void tmpCuda(int* rowPtrU_d, int* colIndU_d, int* rowPtrWVWT_d, int* colIndWVWT_d, int* rowPtrS_d, int num_cam, int* tmp)
{
	unsigned int k = threadIdx.x + (blockDim.x * threadIdx.y) + (blockDim.x * blockDim.y * blockIdx.x);
	unsigned int newvwt, u, up1, wvwt, wvwtp1;
	unsigned int count, nes;

	if (k < (num_cam * 15))
	{
		wvwt = rowPtrWVWT_d[k]; wvwtp1 = rowPtrWVWT_d[k + 1]; newvwt = wvwtp1 - wvwt;
		u = rowPtrU_d[k]; up1 = rowPtrU_d[k + 1]; //neu = up1 - u;
		nes = newvwt;

		for (int i = u; i < up1; i++){
			count = 0;
			for (int j = wvwt; j < wvwtp1; j++){
				if (colIndU_d[i] == colIndWVWT_d[j]){
					count = count + 1;
				}
			}
			if (count == 0){
				nes = nes + 1;
			}
		}
		tmp[k] = nes;
		//printf("values of %d is %d\n", k, nes);
	}
	
	
	__syncthreads();

	
}

__global__ void rowPtrCuda(int* tmp, int* rowPtrS_d, int num_cam)
{
	unsigned int k = threadIdx.x + (blockDim.x * threadIdx.y) + (blockDim.x * blockDim.y * blockIdx.x);
	unsigned int total;

	if (k < (num_cam * 15)){
		
		if (k == 0){
			rowPtrS_d[0] = 0;
		}

		total = 0;
		for (int i = k; i >= 0; i--){
			total = total + tmp[i];
		}
		//printf("values of %d is %d\n", k, total);
		rowPtrS_d[k + 1] = total;
		//printf("values of %d is %d\n", k, rowPtrS_d[k+1]);
	}
	__syncthreads();
}

__global__ void scalCuda(int* rowPtrU_d, int* colIndU_d, dtype* U_d, int* rowPtrWVWT_d, int* colIndWVWT_d, dtype* WVWT_d, int* rowPtrS_d, int* colIndS_d, dtype* S_d, int num_cam)
{
	unsigned int k = threadIdx.x + (blockDim.x * threadIdx.y) + (blockDim.x * blockDim.y * blockIdx.x);
	unsigned int u, up1, wvwt, wvwtp1, s;
	unsigned int newvwt, num = 0;

	if (k < (num_cam * 15)){
		u = rowPtrU_d[k]; up1 = rowPtrU_d[k + 1]; //neu = up1 - u;
		wvwt = rowPtrWVWT_d[k]; wvwtp1 = rowPtrWVWT_d[k + 1]; newvwt = wvwtp1 - wvwt;
		s = rowPtrS_d[k]; //sp1 = rowPtrS_d[k + 1]; //nes = sp1 - s;

		for (int i = u; i < up1; i++){
			for (int j = (wvwt+num); j < wvwtp1; j++){
				if (colIndU_d[i] < colIndWVWT_d[j]){
					colIndS_d[s] = colIndU_d[i];
					S_d[s] = U_d[i];
					s = s + 1;
					break;
				}
				if (colIndU_d[i] == colIndWVWT_d[j]){
					colIndS_d[s] = colIndU_d[i];
					S_d[s] = U_d[i] - WVWT_d[j];
					s = s + 1;
					num = num + 1;
					break;
				}
				if (colIndU_d[i] > colIndWVWT_d[j]){
					colIndS_d[s] = colIndWVWT_d[j];
					S_d[s] = -WVWT_d[j];
					s = s + 1;
					num = num + 1;
				}
			}
		}

		if (num < newvwt){
			for (int i = (wvwt + num); i < wvwtp1; i++){
				colIndS_d[s] = colIndWVWT_d[i];
				S_d[s] = -WVWT_d[i];
				s = s + 1;
			}
		}
		//printf("values of %d is %d\n", k, rowPtrS_d[k + 1]);
	}
	__syncthreads();
}

//__global__ void printCuda(int* row, int size)
__global__ void printCuda(dtype* S_d, int size)
{
	for (int i = 0; i < 100; i++){
		printf("values of %d is %f\n", i, S_d[i]);
	}
}

void sCalculation(dtype* U_d, int* rowPtrU_d, int* colIndU_d, dtype* WVWT_d, int* rowPtrWVWT_d, int* colIndWVWT_d)
{
	hipError_t cudaStat1, cudaStat2, cudaStat3, cudaStat4;
	//First fill out the rowPtrS_d as its size is row(U) + 1
	int* rowPtrS_d, *colIndS_d, *tmp; dtype* S_d;
	cudaStat1 = hipMalloc((void**)&rowPtrS_d, ((num_cam * 15) + 1)*sizeof(rowPtrS_d[0]));
	cudaStat2 = hipMalloc((void**)&tmp, ((num_cam * 15))*sizeof(tmp[0]));
	if ((cudaStat1 == hipSuccess) && (cudaStat2 == hipSuccess)){
		cout << "Successfully initialized rowPtrS_d and tmp" << endl;
	}
	else{
		cout << "****** Initialization of tmp and rowPtrS_d failed" << endl;
	}

	//Call GPU Kernel to fill rowPtrS_d
	int dimx = 16; int dimy = 16;
	cout << "Number of blocks is " << ((num_cam * 15) / (dimx*dimy)) + 1 << endl;
	dim3 block(dimx, dimy); dim3 grid((((num_cam * 15) / (block.x*block.y)) + 1));

	//rowPtrs_d is calculated
	tmpCuda << <grid, block >> >(rowPtrU_d, colIndU_d, rowPtrWVWT_d, colIndWVWT_d, rowPtrS_d, num_cam, tmp);
	hipDeviceSynchronize();
	rowPtrCuda << <grid, block >> >(tmp, rowPtrS_d, num_cam);
	hipDeviceSynchronize();
	hipFree(tmp);

	//Allocate Memory for colIndS_d and S_d
	int jSId, baseS;
	hipMemcpy(&jSId, rowPtrS_d + (num_cam * 15), sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&baseS, rowPtrS_d, sizeof(int), hipMemcpyDeviceToHost);
	jSId -= baseS;

	//cout << "Value of jSId is " << jSId << " baseS is " << baseS << endl;
	cout << "The values of nnzS is " << jSId << " and total are " << (num_cam * num_cam * 225) << endl;

	cudaStat3 = hipMalloc((void**)&colIndS_d, jSId*sizeof(colIndS_d[0]));
	cudaStat4 = hipMalloc((void**)&S_d, jSId*sizeof(S_d[0]));
	if ((cudaStat3 == hipSuccess) && (cudaStat4 == hipSuccess)){
		cout << "Successfully initialized S_d and colIndS_d" << endl;
	}
	else{
		cout << "****** Initialization of S_d and colIndS_d failed" << endl;
	}

	//Calculate U - WVWT
	//Include U in S
	scalCuda << <grid, block >> >(rowPtrU_d, colIndU_d, U_d, rowPtrWVWT_d, colIndWVWT_d, WVWT_d, rowPtrS_d, colIndS_d, S_d,num_cam);
	hipDeviceSynchronize();
	//dim3 block1(1, 1, 1), grid1(1, 1, 1);
	//printCuda << <grid1, block1 >> >(rowPtrS_d,((num_cam*15)+1));
	//printCuda << <grid1, block1 >> >(colIndS_d, jSId);
	//printCuda << <grid1, block1 >> >(S_d, jSId);
	//hipDeviceSynchronize();

	hipFree(U_d);
	hipFree(rowPtrU_d);
	hipFree(colIndU_d);
	hipFree(WVWT_d);
	hipFree(rowPtrWVWT_d);
	hipFree(colIndWVWT_d);

	//Allocate CPU memory for S
	dtype* S = new dtype[jSId];
	int* rowPtrS = new int[(num_cam * 15) + 1];
	int* colIndS = new int[jSId];

	cudaStat1 = hipMemcpy(S, S_d, jSId*sizeof(dtype), hipMemcpyDeviceToHost);
	cudaStat2 = hipMemcpy(rowPtrS, rowPtrS_d, ((num_cam * 15) + 1)*sizeof(int), hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(colIndS, colIndS_d, jSId*sizeof(int), hipMemcpyDeviceToHost);
	if ((cudaStat1 == hipSuccess) && (cudaStat2 == hipSuccess) && (cudaStat3 == hipSuccess)){
		cout << "Successfully copied S into host" << endl;
	}
	else{
		cout << "****** Copy of S into host failed" << endl;
	}

	hipFree(S_d);
	hipFree(rowPtrS_d);
	hipFree(colIndS_d);

	//Print S and verify
	/*for (int i = 0; i < 100; i++){
		cout << rowPtrS[i] << "\t";
	}*/
	cout << endl;
	cout << "************************************************************************" << endl;
	////////////////////////////////////////////////////////////////////////////////////////////////
}